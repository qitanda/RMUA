#include "hip/hip_runtime.h"
#include "superpoint_kernel.cuh"

void nms_kernel(const float *data, int cell, int h, int w, int cell_size, float3 *kps)
{
  int xid = blockDim.x * blockIdx.x + threadIdx.x;
  int yid = blockDim.y * blockIdx.y + threadIdx.y;
  int x = (xid) * cell;
  int y = (yid) * cell;

  float max_prob = 0.0;
  int max_xi = -1, max_yi = -1;
  for(int xi = x; xi < x + cell; xi++) {
    for(int yi = y; yi < y + cell; yi++) {
      if(data[xi + yi * w] > max_prob) {
        max_prob = data[xi + yi * w];
        max_xi = xi;
        max_yi = yi;
      } 
    }
  }
  kps[xid + yid * cell_size] = make_float3(max_xi, max_yi, max_prob);
}

void normalize_descriptor_kernel(float *data)
{

}